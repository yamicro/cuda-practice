#include "hip/hip_runtime.h"
#include "matul.cuh"

#include <base/cuda_config.h>

namespace kernel
{
    template <int THREAD_PER_BLOCK, int ROW_PER_BLOCK>
    __global__ void matmul_kernel_cu_fp32(const float* input, const float* weight, float* C, int M,
                                      int K) {
        __shared__ float sdata[THREAD_PER_BLOCK];
        unsigned int tid = threadIdx.x;

        int start_row = blockIdx.x * ROW_PER_BLOCK;
        int end_row = start_row + ROW_PER_BLOCK;
        if (start_row >= K) {
            return;
        }
        for (int p = start_row; p < end_row; ++p) {
            sdata[tid] = 0;
            for (int i = tid; i < M; i += THREAD_PER_BLOCK) {
                sdata[tid] += input[i] * weight[p * M + i];
            }
            __syncthreads();
            for (unsigned int s = 1; s < THREAD_PER_BLOCK; s *= 2) {
                if ((tid & (2 * s - 1)) == 0) {
                    sdata[tid] += sdata[tid + s];
                }
                __syncthreads();
            }

            if (tid == 0) {
                C[p] = sdata[0];
            }
            __syncthreads();
        }
    }

    void matmul_kernel_cu(const tensor::Tensor& input, const tensor::Tensor& weight,
                          const tensor::Tensor& output, float scale,
                          const CudaConfig* config)
    {
        CHECK(config != nullptr);
        if (config->stream) {
        }
        CHECK(input.is_empty() == false && input.dims_size() <= 2);
        CHECK(input.device_type() == base::DeviceType::kDeviceCUDA);

        CHECK(weight.is_empty() == false && weight.dims_size() == 2);
        CHECK(weight.device_type() == base::DeviceType::kDeviceCUDA);
        const int32_t K = weight.get_dim(0);  // row
        const int32_t M = weight.get_dim(1);  // col
        CHECK_EQ(M, input.get_dim(0));
        matmul_kernel_cu_fp32<256, 1><<<K, 256>>>(input.ptr<float>(), weight.ptr<float>(),
                                                  const_cast<float*>(output.ptr<float>()), M, K);
    }
}
