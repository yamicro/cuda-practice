#include "hip/hip_runtime.h"
#include "rmsnorm_cu.cuh"
#include <cub/block/block_reduce.cuh>
#include <cub/warp/warp_reduce.cuh>
#include <stdio.h>


namespace kernel
{
    static __global__ void row_rmsnorm_f32(const float* &input, const float* weight,
        float* output, const int size, const float eps)
    {
        const int tid = threadIdx.x;
        const int lane_id = tid % warpSize;

        printf("Thread %d: in_ptr[%d] = %f, w_ptr[%d] = %f\n", tid, tid, input[tid], tid, weight[tid]);


        float sum = 0.0f;
        for (int i = lane_id; i < size; i += warpSize)
        {
            sum = input[i] * input[i];
        }

        using WarpReduce = hipcub::WarpReduce<float, 32>;
        __shared__ typename WarpReduce::TempStorage temp;
        __shared__ float shared_val;
        sum = WarpReduce(temp).Reduce(sum, hipcub::Sum());
        if (threadIdx.x == 0) {
            shared_val = sum;
        }
        __syncthreads();
        sum = shared_val;

        const float scale = rsqrtf(sum / static_cast<float>(size) + eps);
        for (int i = lane_id; i < size; i += warpSize) {
            output[i] = scale * input[i] * weight[i];
        }


    }

    void rmsnorm_kernel_cu(const tensor::Tensor& input, const tensor::Tensor& weight,
                           const tensor::Tensor& output, void* stream)
    {
        CHECK(!input.is_empty());
        CHECK(!weight.is_empty());
        CHECK(!output.is_empty());

        CHECK(input.device_type() == base::DeviceType::kDeviceCUDA &&
              weight.device_type() == base::DeviceType::kDeviceCUDA &&
              output.device_type() == base::DeviceType::kDeviceCUDA);

        std::cout << "Input data:" << std::endl;
        for (int i = 0; i < 32 * 15; ++i) {
            std::cout << input.index<float>(i) << " ";
        }
        std::cout << std::endl;

        std::cout << "Weight data:" << std::endl;
        for (int i = 0; i < 32 * 15; ++i) {
            std::cout << weight.index<float>(i) << " ";
        }
        std::cout << std::endl;

        const float eps = 1e-5f;
        const int32_t size = static_cast<int32_t>(input.size());
        const float* in_ptr = input.ptr<float>();
        const float* w_ptr = weight.ptr<float>();
        float* out_ptr = const_cast<float*>(output.ptr<float>());

        if (size < 1024)
        {
            constexpr int threads_num = 128;
            if (stream)
            {
                hipStream_t stream_ = static_cast<hipStream_t>(stream);
                row_rmsnorm_f32<<<1, threads_num, 0, stream_>>>(in_ptr, w_ptr, out_ptr, size, eps);
            }
            else
            {
                row_rmsnorm_f32<<<1, threads_num>>>(in_ptr, w_ptr, out_ptr, size, eps);
            }
        }else {
            constexpr int threads_num = 1024;
            if (stream) {
                hipStream_t stream_ = static_cast<hipStream_t>(stream);
                row_rmsnorm_f32<<<1, threads_num, 0, stream_>>>(in_ptr, w_ptr, out_ptr, size, eps);
            } else {
                row_rmsnorm_f32<<<1, threads_num>>>(in_ptr, w_ptr, out_ptr, size, eps);
            }
        }


    }
}