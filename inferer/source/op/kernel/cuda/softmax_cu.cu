#include "hip/hip_runtime.h"
//
// Created by yami on 25-3-26.
//

#include "softmax_cu.cuh"

#define BLOCK_SIZE 1024

namespace kernel
{
    __global__ void softmax_kernel_cu_fp32(const int64_t total_elements, const int64_t cols, float* input) {
        int row = blockIdx.x;
        int idx = threadIdx.x;
        int offset = row * cols;

        float* row_ptr = input + offset;

        float max_val = -FLT_MAX;
        for (int i = idx; i < cols; i += blockDim.x) {
            max_val = fmaxf(max_val, row_ptr[i]);
        }

        __shared__ float shared_max;
        max_val = blockReduceMax(max_val);
        if (idx == 0) {
            shared_max = max_val;
        }
        __syncthreads();

        float sum_val = 0.0f;
        for (int i = idx; i < cols; i += blockDim.x) {
            sum_val += expf(row_ptr[i] - shared_max);
        }

        __shared__ float shared_sum;
        sum_val = blockReduceSum(sum_val);
        if (idx == 0) {
            shared_sum = sum_val;
        }
        __syncthreads();

        for (int i = idx; i < cols; i += blockDim.x) {
            row_ptr[i] = expf(row_ptr[i] - shared_max) / shared_sum;
        }
    }

    __inline__ __device__ float blockReduceMax(float val) {
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
        }

        __shared__ float shared[32];
        int lane = threadIdx.x % warpSize;
        int warp_id = threadIdx.x / warpSize;

        if (lane == 0) {
            shared[warp_id] = val;
        }
        __syncthreads();

        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : -FLT_MAX;
        if (warp_id == 0) {
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
            }
        }
        return val;
    }

    __inline__ __device__ float blockReduceSum(float val) {
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        }

        __shared__ float shared[32];
        int lane = threadIdx.x % warpSize;
        int warp_id = threadIdx.x / warpSize;

        if (lane == 0) {
            shared[warp_id] = val;
        }
        __syncthreads();

        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0f;
        if (warp_id == 0) {
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                val += __shfl_down_sync(0xFFFFFFFF, val, offset);
            }
        }
        return val;
    }


    void softmax_inplace_cu(const tensor::Tensor& input, void* stream) {
        const int block_size = BLOCK_SIZE;
        int dims = input.dims_size();

        if (dims < 1) {
            printf("Error: Invalid tensor dimension.\n");
            return;
        }

        int64_t rows = 1;
        int64_t cols = input.get_dim(dims - 1);

        for (int i = 0; i < dims - 1; ++i) {
            rows *= input.get_dim(i);
        }

        float* x_ptr = const_cast<float*>(input.ptr<float>());

        dim3 grid(rows);
        dim3 block(block_size);

        hipStream_t stream_ = stream ? static_cast<hipStream_t>(stream) : 0;

        // 启动 kernel 处理 softmax
        softmax_kernel_cu_fp32<<<grid, block, 0, stream_>>>(rows * cols, cols, x_ptr);

        // 错误检查
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }
    }
}

